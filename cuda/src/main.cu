/* 
 * Course: High Performance Computing 2022/2023 
 * 
 * Lecturer: Francesco Moscato  fmoscato@unisa.it 
 *
 * Group:
 * Langella	Antonio   0622702011   a.langella31@studenti.unisa.it
 * Marsico Michele   0622702012   m.marsico10@studenti.unisa.it
 * Paolino Salvatore   0622702016   s.paolino6@studenti.unisa.it 
 * Risi Davide   0622702013   d.risi2@studenti.unisa.it
 * 
 * Copyright (C) 2023 - All Rights Reserved 
 *
 * This file is part of Parallelize-Tarjan-MPI-CUDA. 
 *
 * Parallelize-Tarjan-MPI-CUDA is free software: you can redistribute it and/or modify 
 * it under the terms of the GNU General Public License as published by 
 * the Free Software Foundation, either version 3 of the License, or 
 * (at your option) any later version. 
 *
 * Parallelize-Tarjan-MPI-CUDA is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the 
 * GNU General Public License for more details. 
 *
 * You should have received a copy of the GNU General Public License 
 * along with Parallelize-Tarjan-MPI-CUDA.  If not, see <http://www.gnu.org/licenses/>. 
 * 
 */

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include "cuda_graph.h"
#include "preprocess.cu"
#include "array.h"
#include "graph.h"
#include "measurement.h"

#define THREADxBLOCK 512

using namespace std;

scc_set_t *SCCs;

void callback(array_int * scc){
    int scc_id;
    scc_id = array_int_get_min(scc);
    scc_set_add(SCCs,scc_id,scc);
}

int main(int argc, char **argv){
    char path_inputfilename[100];
    char output_filename[100];
    //for mark cancelled node by cuda
    int *bitmask, *d_bitmask;
    //rappresentazione grafi
    int *d_adj_lists, *d_adj_list_indexes;
    //host
    bool terminate = false;
    //device
    bool *d_terminate;

    int num, thread_number;
    double temp=0.0,time_tarjan = 0.0,time_init = 0.0,time_preprocess=0.0,time_graph_conversion=0.0,time_destroy=0.0;

    if(argc < 3 || argc > 4){
        printf("Error! Wrong or missing parameters. Please run the program specifing the path of the graph to compute and the name the output file.\n");
        exit(1);
    }
    sscanf(argv[1],"%s",path_inputfilename);
    sscanf(argv[2],"%s",output_filename);

    STARTTIME(1);
    SCCs = scc_set_init();

    cuda_graph_t *cuda_graph = cuda_graph_load_from_file(path_inputfilename);
    int n_vertices = cuda_graph->n_vertex;
    int n_bitmask = ((n_vertices-1)/32)+1;
    bitmask = new int[n_bitmask](); //Instantiate an array and initialize it to 0
    //Choose thread number
    if(argc == 3){ //No thread number provided -> thread number equal the number of vertices
      thread_number = n_vertices;
    } else { //Thread number provided as argv[3]
      sscanf(argv[3],"%d",&thread_number);
      if(thread_number < 0){ //Negative thread numbers have a special meaning: -n -> n/4 * n_vertices
        thread_number = ((-thread_number)/4.0) * n_vertices;
      }
    }

    //Eseguire il kernel
    int gridsize = ((thread_number-1)/THREADxBLOCK) + 1; //Numero blocchi su una dimensione della griglia
    dim3 dimGrid(gridsize);
    dim3 dimBlock(THREADxBLOCK);

    hipMalloc(&d_adj_lists, cuda_graph->adj_lists_len * sizeof(int));
    hipMemcpy(d_adj_lists, cuda_graph->adj_lists, cuda_graph->adj_lists_len * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&d_adj_list_indexes, (n_vertices + 1) * sizeof(int));
    hipMemcpy(d_adj_list_indexes, cuda_graph->adj_list_indexes, (n_vertices + 1) * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_bitmask, n_bitmask * sizeof(int));
    hipMemset(d_bitmask, 0, n_bitmask * sizeof(int));

    hipMalloc(&d_terminate, sizeof(bool));
    ENDTIME(1,time_init);
    
    int iter = 0;
    while(!terminate){
        //printf di C++
        //cout << "Called " << iter++ << " times" <<endl;
        terminate = true;
        hipMemset(d_terminate, true, sizeof(bool));
        //printf("start del kernel\n");
        
        STARTTIME(2);
        DeleteTrivialSCCs<<<dimGrid, dimBlock>>>(thread_number, d_adj_lists, d_adj_list_indexes, n_vertices, d_bitmask, d_terminate);
        hipDeviceSynchronize();
        ENDTIME(2,temp);
        time_preprocess += temp;
    
        //hipMemcpy(bitmask, d_bitmask, n_bitmask * sizeof(int), hipMemcpyDeviceToHost);
        /*for(int i = 0; i < n_bitmask; i++){
            printf("%x ", bitmask[i]);
        }
        printf("\n");*/
        //device to host
        hipMemcpy(&terminate, d_terminate, sizeof(bool), hipMemcpyDeviceToHost);
    }
    hipMemcpy(bitmask, d_bitmask, n_bitmask * sizeof(int), hipMemcpyDeviceToHost);
    
    //printf("Bitmask: %x\n", bitmask[0]);
    //First
    //00000000.00000000.00000000.00011000 <-- 31-0
    //00000000.00000000.00000000.00010001 <-- 63-32
    //00000000.00000000.00000000.00000101 <-- 95-64
    //00000000.00000000.00000000.00000101 <-- 127-96
    //Second
    //00000000.00000000.00100001.00011000 <-- 31-0
    //00000000.00000000.00000100.00010011 <-- 63-32
    //00000000.00000000.00000000.00010111 <-- 95-64
    //00000000.00000000.00000000.00000111 <-- 127-96
    //Third
    //00000000.10000000.00100011.00011000 <-- 31-0
    //00000000.00001000.00000100.00010111 <-- 63-32
    //00000000.00000000.00000000.10011111 <-- 95-64
    //00000000.00000000.00000000.00000111 <-- 127-96

    
    STARTTIME(3);
    graph_t* graph = cuda_graph_to_graph(cuda_graph, bitmask);
    ENDTIME(3,time_graph_conversion);

    
    //graph_print_debug(graph);
    
    //printf("start graph\n");
    //graph_print_debug(graph);
    //Lanciare tarjan sequenziale
    STARTTIME(4);
    graph_tarjan_foreach(graph, callback);
    ENDTIME(4,time_tarjan);

    num = graph_get_num_vertex(graph);
    
    STARTTIME(5);
    scc_set_save_to_file(SCCs,output_filename);
    cuda_graph_free(cuda_graph);
    hipFree(d_adj_list_indexes);
    hipFree(d_adj_lists);
    hipFree(d_bitmask);
    hipFree(d_terminate);
    graph_free(graph);
    scc_set_free(SCCs);
    delete[] bitmask;
    ENDTIME(5,time_destroy);
    printf("%d,%f,%f,%f,%f,%f,",num,time_init,time_destroy,time_preprocess,time_graph_conversion,time_tarjan);

    
    return 0;
}