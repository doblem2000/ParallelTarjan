#include "hip/hip_runtime.h"
/* 
 * Course: High Performance Computing 2022/2023 
 * 
 * Lecturer: Francesco Moscato  fmoscato@unisa.it 
 *
 * Group:
 * Langella	Antonio   0622702011   a.langella31@studenti.unisa.it
 * Marsico Michele   0622702012   m.marsico10@studenti.unisa.it
 * Paolino Salvatore   0622702016   s.paolino6@studenti.unisa.it 
 * Risi Davide   0622702013   d.risi2@studenti.unisa.it
 * 
 * Copyright (C) 2023 - All Rights Reserved 
 *
 * This file is part of Parallelize-Tarjan-MPI-CUDA. 
 *
 * Parallelize-Tarjan-MPI-CUDA is free software: you can redistribute it and/or modify 
 * it under the terms of the GNU General Public License as published by 
 * the Free Software Foundation, either version 3 of the License, or 
 * (at your option) any later version. 
 *
 * Parallelize-Tarjan-MPI-CUDA is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the 
 * GNU General Public License for more details. 
 *
 * You should have received a copy of the GNU General Public License 
 * along with Parallelize-Tarjan-MPI-CUDA.  If not, see <http://www.gnu.org/licenses/>. 
 * 
 */

#include <stdio.h>
#include <stdio.h>
#include "bitarray.h"

// kernel uses 10 registers
__global__ void DeleteTrivialSCCs(int* adj_lists, int* adj_list_indexes, int n_vertices, int *bitmask, bool *terminate) {
    int vertex_id = blockDim.x * blockIdx.x + threadIdx.x;
    //If thread is not associated with a vertex or the vertex has already been eliminated then do nothing
    //printf("vertex: %d vertex_id: %d n_vertices: %d\n", vertex_id, vertex_id, n_vertices);
    if(vertex_id >= n_vertices){
        return;
    }

    if(test_bit(bitmask, vertex_id) != 0){ 
        //printf("vertex: %d has already been eliminated\n", vertex_id);
        return;
    }
    int adj_list_start = adj_list_indexes[vertex_id];
    int adj_list_end = adj_list_indexes[vertex_id+1];
    int elim = false;
    //printf("vertex: %d adj_list_start: %d adj_list_end: %d\n", vertex_id, adj_list_start, adj_list_end);
    //If adjacency list is empty or has a self-loop delete the vertex
    if(adj_list_start == adj_list_end){
        elim = true;
    }
    if(((adj_list_start + 1) == adj_list_end) && adj_lists[adj_list_start] == vertex_id){
        elim = true;
    }

    if(elim == true){
        //printf("Sto eliminando1\n");
        (*terminate) = false;
        set_bit(bitmask, vertex_id);
        //printf("vertex: %d eliminated1\n", vertex_id);
        return;
    }

    //If vertex has no incoming edges delete the vertex
    elim = true;
    //printf("adj_list_indexes[n_vertices]%d\n",adj_list_indexes[n_vertices]);
    for(int i = 0; i < adj_list_indexes[n_vertices]; i++){
        if(adj_lists[i] == vertex_id){ //! Manca una condizione
            //printf("vertex: %d found\n", vertex_id);
            elim = false;
            break;
        }
    }

    if(elim == true){
        //printf("vertex: %d eliminated2\n", vertex_id);
        (*terminate) = false;
        set_bit(bitmask, vertex_id);
    }
}