/* 
 * Course: High Performance Computing 2022/2023 
 * 
 * Lecturer: Francesco Moscato  fmoscato@unisa.it 
 *
 * Group:
 * Langella	Antonio   0622702011   a.langella31@studenti.unisa.it
 * Marsico Michele   0622702012   m.marsico10@studenti.unisa.it
 * Paolino Salvatore   0622702016   s.paolino6@studenti.unisa.it 
 * Risi Davide   0622702013   d.risi2@studenti.unisa.it
 * 
 * Copyright (C) 2023 - All Rights Reserved 
 *
 * This file is part of Parallelize-Tarjan-MPI-CUDA. 
 *
 * Parallelize-Tarjan-MPI-CUDA is free software: you can redistribute it and/or modify 
 * it under the terms of the GNU General Public License as published by 
 * the Free Software Foundation, either version 3 of the License, or 
 * (at your option) any later version. 
 *
 * Parallelize-Tarjan-MPI-CUDA is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of 
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the 
 * GNU General Public License for more details. 
 *
 * You should have received a copy of the GNU General Public License 
 * along with Parallelize-Tarjan-MPI-CUDA.  If not, see <http://www.gnu.org/licenses/>. 
 * 
 */

#include <mpi.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <sys/resource.h>
#include "cuda_graph.h"
#include "preprocess.cu"
#include "array.h"
#include "graph.h"
#include "measurement.h"
#include "mpi_logic.h"


#define THREADxBLOCK 1024

using namespace std;

scc_set_t *SCCs;


int main(int argc, char **argv){
    char path_inputfilename[100];
    char output_filename[100];
    int *bitmask, *d_bitmask;
    int *d_adj_lists, *d_adj_list_indexes;
    bool terminate = false;
    bool *d_terminate;

    int rank, size;
    struct rlimit rl;
    getrlimit(RLIMIT_STACK, &rl);
    rl.rlim_cur=128000000;
    setrlimit(RLIMIT_STACK, &rl);
    MPI_Init(&argc,&argv);
    MPI_Comm_size(MPI_COMM_WORLD,&size);
    MPI_Comm_rank(MPI_COMM_WORLD,&rank);

    int num;
    double temp=0.0,time_total_mpi_tarjan = 0.0,time_init = 0.0,time_preprocess=0.0,time_graph_conversion=0.0,time_destroy=0.0;

    if(argc != 3 ){
        printf("Error! Wrong or missing parameters. Please run the program specifing the path of the graph to compute and the name the output file.\n");
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }

    sscanf(argv[1],"%s",path_inputfilename);
    sscanf(argv[2],"%s",output_filename);

    if(size <= 1){
        //Se il numero di processi è 1 allora non posso eseguire il programma in maniera parallela
        printf("This application is meant to be run with at least 2 processes.\n");
        MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
    }

    if(rank == 0){

        STARTTIME(1);
        SCCs = scc_set_init();

        cuda_graph_t *cuda_graph = cuda_graph_load_from_file(path_inputfilename);
        int n_vertices = cuda_graph->n_vertex;
        int n_bitmask = ((n_vertices-1)/32)+1;
        bitmask = new int[n_bitmask](); //Instantiate an array and initialize it to 0
        //Eseguire il kernel
        int gridsize = ((n_vertices-1)/THREADxBLOCK) + 1; //Numero blocchi su una dimensione della griglia
        dim3 dimGrid(gridsize);
        dim3 dimBlock(THREADxBLOCK);

        hipMalloc(&d_adj_lists, cuda_graph->adj_lists_len * sizeof(int));
        hipMemcpy(d_adj_lists, cuda_graph->adj_lists, cuda_graph->adj_lists_len * sizeof(int), hipMemcpyHostToDevice);
        hipMalloc(&d_adj_list_indexes, (n_vertices + 1) * sizeof(int));
        hipMemcpy(d_adj_list_indexes, cuda_graph->adj_list_indexes, (n_vertices + 1) * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&d_bitmask, n_bitmask * sizeof(int));
        hipMemset(d_bitmask, 0, n_bitmask * sizeof(int));

        hipMalloc(&d_terminate, sizeof(bool));
        ENDTIME(1,time_init);
        
        printf("%d,",n_vertices);

        int iter = 0;
        while(!terminate){
            terminate = true;
            hipMemset(d_terminate, true, sizeof(bool));
            
            STARTTIME(2);
            DeleteTrivialSCCs<<<dimGrid, dimBlock>>>(d_adj_lists, d_adj_list_indexes, n_vertices, d_bitmask, d_terminate);
            hipDeviceSynchronize();
            ENDTIME(2,temp);
            time_preprocess += temp;
        
            hipMemcpy(&terminate, d_terminate, sizeof(bool), hipMemcpyDeviceToHost);
        }
        hipMemcpy(bitmask, d_bitmask, n_bitmask * sizeof(int), hipMemcpyDeviceToHost);
        
        //Lanciare tarjan sequenziale
        STARTTIME(3);
        graph_t* graph = cuda_graph_to_graph(cuda_graph, bitmask);
        ENDTIME(3,time_graph_conversion);
        
        STARTTIME(4);
        
        master_work2(rank,size,graph,SCCs,output_filename,time_init);
        
        ENDTIME(4,time_total_mpi_tarjan);

        STARTTIME(5);
        scc_set_save_to_file(SCCs,output_filename);
        cuda_graph_free(cuda_graph);
        hipFree(d_adj_list_indexes);
        hipFree(d_adj_lists);
        hipFree(d_bitmask);
        hipFree(d_terminate);
        graph_free(graph);
        scc_set_free(SCCs);
        delete[] bitmask;
        
        ENDTIME(5,time_destroy);
        printf("%f,%f,%f,%f,",time_total_mpi_tarjan,time_preprocess,time_graph_conversion,time_destroy);

    }

    if(rank != 0){
        slave_work(rank);
    }

    MPI_Finalize();
    return 0;
}